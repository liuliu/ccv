#include "hip/hip_runtime.h"
extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

#ifdef HAVE_CUDA

template<typename NUM>
__global__ void _ccv_nnc_upsample_nearest_forw_nchw(const int hw, const float rwidth, const float rheight, const int nc, const int adim2, const int ainc2, const int adim3, const int ainc3, const NUM* const a, const int binc2, const int bdim3, const int binc3, NUM* const b)
{
	CUDA_1D_KERNEL_LOOP(i, hw) {
		const int xd = i % bdim3;
		const int yd = i / bdim3;
		const NUM* ap = a;
		NUM* bp = b;
		const float xs = (xd + 0.5) * rwidth;
		const int xsi0 = ccv_min((int)xs, adim3 - 1);
		const float ys = (yd + 0.5) * rheight;
		const int ysi0 = ccv_min((int)ys, adim2 - 1);
		for (int j = 0; j < nc; j++)
		{
			bp[xd + yd * binc3] = (NUM)ap[xsi0 + ysi0 * ainc3];
			ap += ainc2;
			bp += binc2;
		}
	}
}

template<typename NUM>
__global__ void _ccv_nnc_upsample_nearest_forw_nhwc(const int hw, const float rwidth, const float rheight, const int n, const int c, const int adim1, const int ainc1, const int adim2, const int ainc2, const int ainc3, const NUM* const a, const int binc1, const int bdim2, const int binc2, const int binc3, NUM* const b)
{
	CUDA_1D_KERNEL_LOOP(i, hw) {
		const int xd = i % bdim2;
		const int yd = i / bdim2;
		const NUM* ap = a;
		NUM* bp = b;
		const float xs = (xd + 0.5) * rwidth;
		const int xsi0 = ccv_min((int)xs, adim2 - 1);
		const float ys = (yd + 0.5) * rheight;
		const int ysi0 = ccv_min((int)ys, adim1 - 1);
		for (int j = 0; j < n; j++)
		{
			for (int k = 0; k < c; k++)
				bp[k + xd * binc3 + yd * binc2] = (NUM)ap[k + xsi0 * ainc3 + ysi0 * ainc2];
			ap += ainc1;
			bp += binc1;
		}
	}
}

template<typename NUM>
__global__ void _ccv_nnc_zero_back(const size_t tensor_count, NUM* const a)
{
	CUDA_1D_KERNEL_LOOP(i, tensor_count) {
		a[i] = 0;
	}
}

template<typename NUM>
__global__ void _ccv_nnc_upsample_nearest_back_nchw(const size_t tensor_count, const float rwidth, const float rheight, const int adim2, const int ainc2, const int adim3, const int ainc3, NUM* const a, const int bdim2, const int binc2, const int bdim3, const int binc3, const NUM* const b)
{
	CUDA_1D_KERNEL_LOOP(i, tensor_count) {
		const int xd = i % bdim3;
		const int idxyd = i / bdim3;
		const int yd = idxyd % bdim2;
		const int idx = idxyd / bdim2;
		NUM* const ap = a + idx * ainc2;
		const NUM* const bp = b + idx * binc2;
		const float xs = (xd + 0.5) * rwidth;
		const int xsi0 = ccv_min((int)xs, adim3 - 1);
		const float ys = (yd + 0.5) * rheight;
		const int ysi0 = ccv_min((int)ys, adim2 - 1);
		const float bpi = (float)__ldg(bp + xd + yd * binc3);
		atomicAdd(&ap[xsi0 + ysi0 * ainc3], (NUM)bpi);
	}
}

template<typename NUM>
__global__ void _ccv_nnc_upsample_nearest_back_nhwc(const size_t tensor_count, const float rwidth, const float rheight, const int ch, const int adim1, const int ainc1, const int adim2, const int ainc2, const int ainc3, NUM* const a, const int bdim1, const int binc1, const int bdim2, const int binc2, const int binc3, const NUM* const b)
{
	CUDA_1D_KERNEL_LOOP(i, tensor_count) {
		const int xd = i % bdim2;
		const int idxyd = i / bdim2;
		const int yd = idxyd % bdim1;
		const int idx = idxyd / bdim1;
		NUM* const ap = a + idx * ainc1;
		const NUM* const bp = b + idx * binc1;
		const float xs = (xd + 0.5) * rwidth;
		const int xsi0 = ccv_min((int)xs, adim2 - 1);
		const float ys = (yd + 0.5) * rheight;
		const int ysi0 = ccv_min((int)ys, adim1 - 1);
		for (int c = 0; c < ch; c++)
		{
			const float bpi = (float)__ldg(bp + c + xd * binc3 + yd * binc2);
			atomicAdd(&ap[c + xsi0 * ainc3 + ysi0 * ainc2], (NUM)bpi);
		}
	}
}

static int _ccv_nnc_upsample_nearest_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size >= 1);
	assert(output_size >= 1);
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	ccv_nnc_tensor_view_t* const a = (ccv_nnc_tensor_view_t*)inputs[0];
	ccv_nnc_tensor_view_t* const b = (ccv_nnc_tensor_view_t*)outputs[0];
	assert(ccv_nnc_tensor_nd(a->info.dim) <= CCV_NNC_MAX_DIM + 2);
	assert(ccv_nnc_tensor_nd(b->info.dim) <= CCV_NNC_MAX_DIM + 2);
	// Assuming this is float 32.
	int adim[CCV_NNC_MAX_DIM_ALLOC];
	int bdim[CCV_NNC_MAX_DIM_ALLOC];
	ccv_nnc_tensor_view_get_dim(a, adim);
	ccv_nnc_tensor_view_get_dim(b, bdim);
	int astride[CCV_NNC_MAX_DIM_ALLOC];
	int bstride[CCV_NNC_MAX_DIM_ALLOC];
	assert(CCV_NNC_MAX_DIM == 2); // Need to change this logic for CCV_NNC_MAX_DIM == other number.
	ccv_nnc_tensor_view_get_stride(a, astride);
	ccv_nnc_tensor_view_get_stride(b, bstride);
	assert(a->info.format == b->info.format);
	assert(a->info.datatype == b->info.datatype);
	if (a->info.format == CCV_TENSOR_FORMAT_NCHW)
	{
		assert(adim[0] == bdim[0]);
		assert(adim[1] == bdim[1]);
		const int hw = bdim[2] * bdim[3];
		const float rheight = (float)adim[2] / bdim[2];
		const float rwidth = (float)adim[3] / bdim[3];
		assert(rheight <= 1);
		assert(rwidth <= 1);
		if (a->info.datatype == CCV_32F)
			_ccv_nnc_upsample_nearest_forw_nchw<<<CUDA_GET_BLOCKS(hw), CUDA_NUM_THREADS, 0, stream>>>(hw, rwidth, rheight, adim[0] * adim[1], adim[2], astride[1], adim[3], astride[2], a->data.f32, bstride[1], bdim[3], bstride[2], b->data.f32);
		else if (a->info.datatype == CCV_16F)
			_ccv_nnc_upsample_nearest_forw_nchw<<<CUDA_GET_BLOCKS(hw), CUDA_NUM_THREADS, 0, stream>>>(hw, rwidth, rheight, adim[0] * adim[1], adim[2], astride[1], adim[3], astride[2], (__half*)a->data.f16, bstride[1], bdim[3], bstride[2], (__half*)b->data.f16);
	} else {
		assert(a->info.format == CCV_TENSOR_FORMAT_NHWC || a->info.format == CCV_TENSOR_FORMAT_CHWN);
		const float rheight = (float)adim[1] / bdim[1];
		const float rwidth = (float)adim[2] / bdim[2];
		assert(rheight <= 1);
		assert(rwidth <= 1);
		const int hw = bdim[1] * bdim[2];
		if (a->info.datatype == CCV_32F)
			_ccv_nnc_upsample_nearest_forw_nhwc<<<CUDA_GET_BLOCKS(hw), CUDA_NUM_THREADS, 0, stream>>>(hw, rwidth, rheight, adim[0], adim[3], adim[1], astride[0], adim[2], astride[1], astride[2], a->data.f32, bstride[0], bdim[2], bstride[1], bstride[2], b->data.f32);
		else if (a->info.datatype == CCV_16F)
			_ccv_nnc_upsample_nearest_forw_nhwc<<<CUDA_GET_BLOCKS(hw), CUDA_NUM_THREADS, 0, stream>>>(hw, rwidth, rheight, adim[0], adim[3], adim[1], astride[0], adim[2], astride[1], astride[2], (__half*)a->data.f16, bstride[0], bdim[2], bstride[1], bstride[2], (__half*)b->data.f16);
	}
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_upsample_nearest_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size >= 1);
	assert(output_size >= 1);
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	ccv_nnc_tensor_view_t* const a = (ccv_nnc_tensor_view_t*)outputs[0];
	ccv_nnc_tensor_view_t* const b = (ccv_nnc_tensor_view_t*)inputs[0];
	assert(ccv_nnc_tensor_nd(a->info.dim) <= CCV_NNC_MAX_DIM + 2);
	assert(ccv_nnc_tensor_nd(b->info.dim) <= CCV_NNC_MAX_DIM + 2);
	// Assuming this is float 32.
	int adim[CCV_NNC_MAX_DIM_ALLOC];
	int bdim[CCV_NNC_MAX_DIM_ALLOC];
	ccv_nnc_tensor_view_get_dim(a, adim);
	ccv_nnc_tensor_view_get_dim(b, bdim);
	int astride[CCV_NNC_MAX_DIM_ALLOC];
	int bstride[CCV_NNC_MAX_DIM_ALLOC];
	assert(CCV_NNC_MAX_DIM == 2); // Need to change this logic for CCV_NNC_MAX_DIM == other number.
	ccv_nnc_tensor_view_get_stride(a, astride);
	ccv_nnc_tensor_view_get_stride(b, bstride);
	assert(a->info.format == b->info.format);
	assert(a->info.datatype == b->info.datatype);
	const size_t a_tensor_count = ccv_nnc_tensor_count(a->info);
	_ccv_nnc_zero_back<<<CUDA_GET_BLOCKS(a_tensor_count), CUDA_NUM_THREADS, 0, stream>>>(a_tensor_count, a->data.f32);
	if (a->info.format == CCV_TENSOR_FORMAT_NCHW)
	{
		const size_t tensor_count = ccv_nnc_tensor_count(b->info);
		const float rheight = (float)adim[2] / bdim[2];
		const float rwidth = (float)adim[3] / bdim[3];
		assert(rheight <= 1);
		assert(rwidth <= 1);
		if (a->info.datatype == CCV_32F)
			_ccv_nnc_upsample_nearest_back_nchw<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, rwidth, rheight, adim[2], astride[1], adim[3], astride[2], a->data.f32, bdim[2], bstride[1], bdim[3], bstride[2], b->data.f32);
		else if (a->info.datatype == CCV_16F)
			_ccv_nnc_upsample_nearest_back_nchw<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, rwidth, rheight, adim[2], astride[1], adim[3], astride[2], (__half*)a->data.f16, bdim[2], bstride[1], bdim[3], bstride[2], (__half*)b->data.f16);
	} else {
		assert(a->info.format == CCV_TENSOR_FORMAT_NHWC || a->info.format == CCV_TENSOR_FORMAT_CHWN);
		const float rheight = (float)adim[1] / bdim[1];
		const float rwidth = (float)adim[2] / bdim[2];
		assert(rheight <= 1);
		assert(rwidth <= 1);
		const size_t tensor_count = ccv_nnc_tensor_count(b->info) / adim[3];
		if (a->info.datatype == CCV_32F)
			_ccv_nnc_upsample_nearest_back_nhwc<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, rwidth, rheight, adim[3], adim[1], astride[0], adim[2], astride[1], astride[2], a->data.f32, bdim[1], bstride[0], bdim[2], bstride[1], bstride[2], b->data.f32);
		else if (a->info.datatype == CCV_16F)
			_ccv_nnc_upsample_nearest_back_nhwc<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, rwidth, rheight, adim[3], adim[1], astride[0], adim[2], astride[1], astride[2], (__half*)a->data.f16, bdim[1], bstride[0], bdim[2], bstride[1], bstride[2], (__half*)b->data.f16);
	}
	return CCV_NNC_EXEC_SUCCESS;
}

template<typename NUM>
__global__ void _ccv_nnc_upsample_bilinear_forw_nchw(const int hw, const float rwidth, const float rheight, const int nc, const int adim2, const int ainc2, const int adim3, const int ainc3, const NUM* const a, const int binc2, const int bdim3, const int binc3, NUM* const b)
{
	CUDA_1D_KERNEL_LOOP(i, hw) {
		const int xd = i % bdim3;
		const int yd = i / bdim3;
		const NUM* ap = a;
		NUM* bp = b;
		const float xs = (xd + 0.5) * rwidth - 0.5;
		const int xsi0 = (int)xs;
		const int xsi1 = ccv_min((int)(xs + 1), adim3 - 1);
		const float xsc1 = xs - xsi0;
		const float xsc0 = 1.0 - xsc1;
		const float ys = (yd + 0.5) * rheight - 0.5;
		const int ysi0 = (int)ys;
		const int ysi1 = ccv_min((int)(ys + 1), adim2 - 1);
		const float ysc1 = ys - ysi0;
		const float ysc0 = 1.0 - ysc1;
		for (int j = 0; j < nc; j++)
		{
			bp[xd + yd * binc3] = (NUM)((float)ap[xsi0 + ysi0 * ainc3] * xsc0 * ysc0 + (float)ap[xsi1 + ysi0 * ainc3] * xsc1 * ysc0 + (float)ap[xsi0 + ysi1 * ainc3] * xsc0 * ysc1 + (float)ap[xsi1 + ysi1 * ainc3] * xsc1 * ysc1);
			ap += ainc2;
			bp += binc2;
		}
	}
}

template<typename NUM>
__global__ void _ccv_nnc_upsample_bilinear_forw_nhwc(const int hw, const float rwidth, const float rheight, const int n, const int c, const int adim1, const int ainc1, const int adim2, const int ainc2, const int ainc3, const NUM* const a, const int binc1, const int bdim2, const int binc2, const int binc3, NUM* const b)
{
	CUDA_1D_KERNEL_LOOP(i, hw) {
		const int xd = i % bdim2;
		const int yd = i / bdim2;
		const NUM* ap = a;
		NUM* bp = b;
		const float xs = (xd + 0.5) * rwidth - 0.5;
		const int xsi0 = (int)xs;
		const int xsi1 = ccv_min((int)(xs + 1), adim2 - 1);
		const float xsc1 = xs - xsi0;
		const float xsc0 = 1.0 - xsc1;
		const float ys = (yd + 0.5) * rheight - 0.5;
		const int ysi0 = (int)ys;
		const int ysi1 = ccv_min((int)(ys + 1), adim1 - 1);
		const float ysc1 = ys - ysi0;
		const float ysc0 = 1.0 - ysc1;
		for (int j = 0; j < n; j++)
		{
			for (int k = 0; k < c; k++)
				bp[k + xd * binc3 + yd * binc2] = (NUM)((float)ap[k + xsi0 * ainc3 + ysi0 * ainc2] * xsc0 * ysc0 + (float)ap[k + xsi1 * ainc3 + ysi0 * ainc2] * xsc1 * ysc0 + (float)ap[k + xsi0 * ainc3 + ysi1 * ainc2] * xsc0 * ysc1 + (float)ap[k + xsi1 * ainc3 + ysi1 * ainc2] * xsc1 * ysc1);
			ap += ainc1;
			bp += binc1;
		}
	}
}

template<typename NUM>
__global__ void _ccv_nnc_upsample_bilinear_back_nchw(const size_t tensor_count, const float rwidth, const float rheight, const int adim2, const int ainc2, const int adim3, const int ainc3, NUM* const a, const int bdim2, const int binc2, const int bdim3, const int binc3, const NUM* const b)
{
	CUDA_1D_KERNEL_LOOP(i, tensor_count) {
		const int xd = i % bdim3;
		const int idxyd = i / bdim3;
		const int yd = idxyd % bdim2;
		const int idx = idxyd / bdim2;
		NUM* const ap = a + idx * ainc2;
		const NUM* const bp = b + idx * binc2;
		const float xs = (xd + 0.5) * rwidth - 0.5;
		const int xsi0 = (int)xs;
		const int xsi1 = ccv_min((int)(xs + 1), adim3 - 1);
		const float xsc1 = xs - xsi0;
		const float xsc0 = 1.0 - xsc1;
		const float ys = (yd + 0.5) * rheight - 0.5;
		const int ysi0 = (int)ys;
		const int ysi1 = ccv_min((int)(ys + 1), adim2 - 1);
		const float ysc1 = ys - ysi0;
		const float ysc0 = 1.0 - ysc1;
		const float bpi = (float)__ldg(bp + xd + yd * binc3);
		atomicAdd(&ap[xsi0 + ysi0 * ainc3], (NUM)(bpi * xsc0 * ysc0));
		atomicAdd(&ap[xsi1 + ysi0 * ainc3], (NUM)(bpi * xsc1 * ysc0));
		atomicAdd(&ap[xsi0 + ysi1 * ainc3], (NUM)(bpi * xsc0 * ysc1));
		atomicAdd(&ap[xsi1 + ysi1 * ainc3], (NUM)(bpi * xsc1 * ysc1));
	}
}

template<typename NUM>
__global__ void _ccv_nnc_upsample_bilinear_back_nhwc(const size_t tensor_count, const float rwidth, const float rheight, const int ch, const int adim1, const int ainc1, const int adim2, const int ainc2, const int ainc3, NUM* const a, const int bdim1, const int binc1, const int bdim2, const int binc2, const int binc3, const NUM* const b)
{
	CUDA_1D_KERNEL_LOOP(i, tensor_count) {
		const int xd = i % bdim2;
		const int idxyd = i / bdim2;
		const int yd = idxyd % bdim1;
		const int idx = idxyd / bdim1;
		NUM* const ap = a + idx * ainc1;
		const NUM* const bp = b + idx * binc1;
		const float xs = (xd + 0.5) * rwidth - 0.5;
		const int xsi0 = (int)xs;
		const int xsi1 = ccv_min((int)(xs + 1), adim2 - 1);
		const float xsc1 = xs - xsi0;
		const float xsc0 = 1.0 - xsc1;
		const float ys = (yd + 0.5) * rheight - 0.5;
		const int ysi0 = (int)ys;
		const int ysi1 = ccv_min((int)(ys + 1), adim1 - 1);
		const float ysc1 = ys - ysi0;
		const float ysc0 = 1.0 - ysc1;
		for (int c = 0; c < ch; c++)
		{
			const float bpi = (float)__ldg(bp + c + xd * binc3 + yd * binc2);
			atomicAdd(&ap[c + xsi0 * ainc3 + ysi0 * ainc2], (NUM)(bpi * xsc0 * ysc0));
			atomicAdd(&ap[c + xsi1 * ainc3 + ysi0 * ainc2], (NUM)(bpi * xsc1 * ysc0));
			atomicAdd(&ap[c + xsi0 * ainc3 + ysi1 * ainc2], (NUM)(bpi * xsc0 * ysc1));
			atomicAdd(&ap[c + xsi1 * ainc3 + ysi1 * ainc2], (NUM)(bpi * xsc1 * ysc1));
		}
	}
}

static int _ccv_nnc_upsample_bilinear_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size >= 1);
	assert(output_size >= 1);
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	ccv_nnc_tensor_view_t* const a = (ccv_nnc_tensor_view_t*)inputs[0];
	ccv_nnc_tensor_view_t* const b = (ccv_nnc_tensor_view_t*)outputs[0];
	assert(ccv_nnc_tensor_nd(a->info.dim) <= CCV_NNC_MAX_DIM + 2);
	assert(ccv_nnc_tensor_nd(b->info.dim) <= CCV_NNC_MAX_DIM + 2);
	// Assuming this is float 32.
	int adim[CCV_NNC_MAX_DIM_ALLOC];
	int bdim[CCV_NNC_MAX_DIM_ALLOC];
	ccv_nnc_tensor_view_get_dim(a, adim);
	ccv_nnc_tensor_view_get_dim(b, bdim);
	int astride[CCV_NNC_MAX_DIM_ALLOC];
	int bstride[CCV_NNC_MAX_DIM_ALLOC];
	assert(CCV_NNC_MAX_DIM == 2); // Need to change this logic for CCV_NNC_MAX_DIM == other number.
	ccv_nnc_tensor_view_get_stride(a, astride);
	ccv_nnc_tensor_view_get_stride(b, bstride);
	assert(a->info.format == b->info.format);
	assert(a->info.datatype == b->info.datatype);
	if (a->info.format == CCV_TENSOR_FORMAT_NCHW)
	{
		assert(adim[0] == bdim[0]);
		assert(adim[1] == bdim[1]);
		const int hw = bdim[2] * bdim[3];
		const float rheight = (float)adim[2] / bdim[2];
		const float rwidth = (float)adim[3] / bdim[3];
		assert(rheight <= 1);
		assert(rwidth <= 1);
		if (a->info.datatype == CCV_32F)
			_ccv_nnc_upsample_bilinear_forw_nchw<<<CUDA_GET_BLOCKS(hw), CUDA_NUM_THREADS, 0, stream>>>(hw, rwidth, rheight, adim[0] * adim[1], adim[2], astride[1], adim[3], astride[2], a->data.f32, bstride[1], bdim[3], bstride[2], b->data.f32);
		else if (a->info.datatype == CCV_16F)
			_ccv_nnc_upsample_bilinear_forw_nchw<<<CUDA_GET_BLOCKS(hw), CUDA_NUM_THREADS, 0, stream>>>(hw, rwidth, rheight, adim[0] * adim[1], adim[2], astride[1], adim[3], astride[2], (__half*)a->data.f16, bstride[1], bdim[3], bstride[2], (__half*)b->data.f16);
	} else {
		assert(a->info.format == CCV_TENSOR_FORMAT_NHWC || a->info.format == CCV_TENSOR_FORMAT_CHWN);
		const float rheight = (float)adim[1] / bdim[1];
		const float rwidth = (float)adim[2] / bdim[2];
		assert(rheight <= 1);
		assert(rwidth <= 1);
		const int hw = bdim[1] * bdim[2];
		if (a->info.datatype == CCV_32F)
			_ccv_nnc_upsample_bilinear_forw_nhwc<<<CUDA_GET_BLOCKS(hw), CUDA_NUM_THREADS, 0, stream>>>(hw, rwidth, rheight, adim[0], adim[3], adim[1], astride[0], adim[2], astride[1], astride[2], a->data.f32, bstride[0], bdim[2], bstride[1], bstride[2], b->data.f32);
		else if (a->info.datatype == CCV_16F)
			_ccv_nnc_upsample_bilinear_forw_nhwc<<<CUDA_GET_BLOCKS(hw), CUDA_NUM_THREADS, 0, stream>>>(hw, rwidth, rheight, adim[0], adim[3], adim[1], astride[0], adim[2], astride[1], astride[2], (__half*)a->data.f16, bstride[0], bdim[2], bstride[1], bstride[2], (__half*)b->data.f16);
	}
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_upsample_bilinear_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size >= 1);
	assert(output_size >= 1);
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	ccv_nnc_tensor_view_t* const a = (ccv_nnc_tensor_view_t*)outputs[0];
	ccv_nnc_tensor_view_t* const b = (ccv_nnc_tensor_view_t*)inputs[0];
	assert(ccv_nnc_tensor_nd(a->info.dim) <= CCV_NNC_MAX_DIM + 2);
	assert(ccv_nnc_tensor_nd(b->info.dim) <= CCV_NNC_MAX_DIM + 2);
	// Assuming this is float 32.
	int adim[CCV_NNC_MAX_DIM_ALLOC];
	int bdim[CCV_NNC_MAX_DIM_ALLOC];
	ccv_nnc_tensor_view_get_dim(a, adim);
	ccv_nnc_tensor_view_get_dim(b, bdim);
	int astride[CCV_NNC_MAX_DIM_ALLOC];
	int bstride[CCV_NNC_MAX_DIM_ALLOC];
	assert(CCV_NNC_MAX_DIM == 2); // Need to change this logic for CCV_NNC_MAX_DIM == other number.
	ccv_nnc_tensor_view_get_stride(a, astride);
	ccv_nnc_tensor_view_get_stride(b, bstride);
	assert(a->info.format == b->info.format);
	assert(a->info.datatype == b->info.datatype);
	const size_t a_tensor_count = ccv_nnc_tensor_count(a->info);
	_ccv_nnc_zero_back<<<CUDA_GET_BLOCKS(a_tensor_count), CUDA_NUM_THREADS, 0, stream>>>(a_tensor_count, a->data.f32);
	if (a->info.format == CCV_TENSOR_FORMAT_NCHW)
	{
		const size_t tensor_count = ccv_nnc_tensor_count(b->info);
		const float rheight = (float)adim[2] / bdim[2];
		const float rwidth = (float)adim[3] / bdim[3];
		assert(rheight <= 1);
		assert(rwidth <= 1);
		if (a->info.datatype == CCV_32F)
			_ccv_nnc_upsample_bilinear_back_nchw<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, rwidth, rheight, adim[2], astride[1], adim[3], astride[2], a->data.f32, bdim[2], bstride[1], bdim[3], bstride[2], b->data.f32);
		else if (a->info.datatype == CCV_16F)
			_ccv_nnc_upsample_bilinear_back_nchw<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, rwidth, rheight, adim[2], astride[1], adim[3], astride[2], (__half*)a->data.f16, bdim[2], bstride[1], bdim[3], bstride[2], (__half*)b->data.f16);
	} else {
		assert(a->info.format == CCV_TENSOR_FORMAT_NHWC || a->info.format == CCV_TENSOR_FORMAT_CHWN);
		const float rheight = (float)adim[1] / bdim[1];
		const float rwidth = (float)adim[2] / bdim[2];
		assert(rheight <= 1);
		assert(rwidth <= 1);
		const size_t tensor_count = ccv_nnc_tensor_count(b->info) / adim[3];
		if (a->info.datatype == CCV_32F)
			_ccv_nnc_upsample_bilinear_back_nhwc<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, rwidth, rheight, adim[3], adim[1], astride[0], adim[2], astride[1], astride[2], a->data.f32, bdim[1], bstride[0], bdim[2], bstride[1], bstride[2], b->data.f32);
		else if (a->info.datatype == CCV_16F)
			_ccv_nnc_upsample_bilinear_back_nhwc<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, rwidth, rheight, adim[3], adim[1], astride[0], adim[2], astride[1], astride[2], (__half*)a->data.f16, bdim[1], bstride[0], bdim[2], bstride[1], bstride[2], (__half*)b->data.f16);
	}
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_upsample_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	if (cmd.info.upsample.type == CCV_NNC_UPSAMPLE_NEAREST)
		return _ccv_nnc_upsample_nearest_forw(cmd, hint, flags, inputs, input_size, outputs, output_size, stream_context);
	else if (cmd.info.upsample.type == CCV_NNC_UPSAMPLE_BILINEAR)
		return _ccv_nnc_upsample_bilinear_forw(cmd, hint, flags, inputs, input_size, outputs, output_size, stream_context);
	return CCV_NNC_EXEC_INVALID;
}

static int _ccv_nnc_upsample_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	if (cmd.info.upsample.type == CCV_NNC_UPSAMPLE_NEAREST)
		return _ccv_nnc_upsample_nearest_back(cmd, hint, flags, inputs, input_size, outputs, output_size, stream_context);
	else if (cmd.info.upsample.type == CCV_NNC_UPSAMPLE_BILINEAR)
		return _ccv_nnc_upsample_bilinear_back(cmd, hint, flags, inputs, input_size, outputs, output_size, stream_context);
	return CCV_NNC_EXEC_INVALID;
}

#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_UPSAMPLE_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDA
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_upsample_forw;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_UPSAMPLE_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDA
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_upsample_back;
#endif
}
