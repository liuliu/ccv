#include "hip/hip_runtime.h"
extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

#ifdef HAVE_CUDNN

__global__ void _ccv_nnc_isnan_kernel(const size_t tensor_count, const float* const a, int* const b)
{
	CUDA_1D_KERNEL_LOOP(i, tensor_count) {
		b[i] = isnan(a[i]) ? 1 : 0;
	}
}

static int _ccv_nnc_reduce_isnan_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 1);
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	ccv_nnc_tensor_view_t atv = ccv_nnc_get_tensor_view(inputs[0]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
	ccv_nnc_tensor_view_t btv = ccv_nnc_get_tensor_view(outputs[0]);
	btv.info.datatype = CCV_32F; // First set it to 32F so it can be properly computed. Then we will turn it into 32S.
	ccv_nnc_tensor_view_t* tvs[] = {
		&atv, &btv
	};
	ccv_nnc_tensor_view_alignment(tvs, 2);
	const ccv_nnc_cudnn_tensor_view_descriptor_t b = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, &btv);
	int can_reduce = 0;
	int i;
	for (i = 0; !can_reduce && i < cmd.info.reduce.count; i++)
		can_reduce = (inputs[0]->info.dim[cmd.info.reduce.axis[i]] > 1);
	static const float one = 1, zero = 0;
	const size_t tensor_count = ccv_nnc_tensor_count(outputs[0]->info);
	if (can_reduce)
	{
		hipdnnReduceTensorDescriptor_t reduce_max = ccv_nnc_stream_context_get_reduce_tensor_descriptor(stream_context);
		hipdnnSetReduceTensorDescriptor(reduce_max, HIPDNN_REDUCE_TENSOR_MAX, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN, HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES);
		void* workspace = 0;
		size_t workspace_size = 0;
		CUDNN_ENFORCE(hipdnnGetReductionWorkspaceSize(cudnn, reduce_max, a.descriptor, b.descriptor, &workspace_size));
		if (workspace_size)
		{
			workspace = ccv_nnc_stream_context_get_workspace(stream_context, workspace_size, CCV_TENSOR_GPU_MEMORY);
			assert(workspace);
		}
		CUDNN_ENFORCE(hipdnnReduceTensor(cudnn, reduce_max, 0, 0, workspace, workspace_size, &one, a.descriptor, a.data.u8, &zero, b.descriptor, b.data.u8));
		ccv_nnc_stream_context_return_reduce_tensor_descriptor(stream_context, reduce_max);
	} else if (a.data.u8 != b.data.u8) {
		// Don't need to reduce, just transfer to b, if the pointer doesn't match.
		CUDNN_ENFORCE(cudnnTransformTensor(cudnn, &one, a.descriptor, a.data.u8,  &zero, b.descriptor, b.data.u8));
	}
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(b);
	_ccv_nnc_isnan_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, outputs[0]->data.f32, outputs[0]->data.i32);
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_reduce_isnan_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	return CCV_NNC_EXEC_INVALID;
}

#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_REDUCE_ISNAN_FORWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F | CCV_32S;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_reduce_isnan_forw;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_REDUCE_ISNAN_BACKWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F | CCV_32S;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_reduce_isnan_back;
#endif
}

