#include "hip/hip_runtime.h"
extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

#ifdef HAVE_CUDA

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_adamw_kernel(const size_t tensor_count, const float scale, const float beta1, const float beta2, const float rate_decay, const float rate_inv_bias_correction1, const float inv_bias_correction2, const float epsilon, const NUM1* const g, const NUM2* const a, const NUM2* const mom, const NUM2* const vel, NUM2* const b, NUM2* const new_mom, NUM2* const new_vel)
{
	CUDA_1D_KERNEL_LOOP(i, tensor_count) {
		const float grad = scale * (float)g[i];
		const float m = beta1 * (float)mom[i] + (1 - beta1) * grad;
		const float v = beta2 * (float)vel[i] + (1 - beta2) * grad * grad;
		b[i] = (NUM2)((float)a[i] - rate_decay * (float)a[i] - (m * rate_inv_bias_correction1) / (sqrtf(v * inv_bias_correction2) + epsilon));
		new_mom[i] = (NUM2)m;
		new_vel[i] = (NUM2)v;
	}
}

static int _ccv_nnc_adamw_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 4);
	assert(output_size == 3);
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	const int step = cmd.info.adam.step;
	const float rate = cmd.info.adam.rate;
	const float scale = cmd.info.adam.scale;
	const float beta1 = cmd.info.adam.beta1;
	const float beta2 = cmd.info.adam.beta2;
	const float decay = cmd.info.adam.decay;
	const float epsilon = cmd.info.adam.epsilon;
	assert(step >= 1);
	const float rate_inv_bias_correction1 = rate / (1 - powf(beta1, step));
	const float inv_bias_correction2 = 1. / (1 - powf(beta2, step));
	const float rate_decay = rate * decay;
	assert(inputs[1]->info.datatype == inputs[2]->info.datatype &&
		inputs[2]->info.datatype == inputs[3]->info.datatype &&
		inputs[3]->info.datatype == outputs[0]->info.datatype &&
		outputs[0]->info.datatype == outputs[1]->info.datatype &&
		outputs[1]->info.datatype == outputs[2]->info.datatype);
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[0]));
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[1]));
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[2]));
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[3]));
	assert(CCV_IS_TENSOR_CONTIGUOUS(outputs[0]));
	assert(CCV_IS_TENSOR_CONTIGUOUS(outputs[1]));
	assert(CCV_IS_TENSOR_CONTIGUOUS(outputs[2]));
	const ccv_nnc_tensor_view_t* const g = (ccv_nnc_tensor_view_t*)inputs[0];
	const ccv_nnc_tensor_view_t* const a = (ccv_nnc_tensor_view_t*)inputs[1];
	const ccv_nnc_tensor_view_t* const m = (ccv_nnc_tensor_view_t*)inputs[2];
	const ccv_nnc_tensor_view_t* const v = (ccv_nnc_tensor_view_t*)inputs[3];
	ccv_nnc_tensor_view_t* const b = (ccv_nnc_tensor_view_t*)outputs[0];
	ccv_nnc_tensor_view_t* const n = (ccv_nnc_tensor_view_t*)outputs[1];
	ccv_nnc_tensor_view_t* const u = (ccv_nnc_tensor_view_t*)outputs[2];
	const size_t tensor_count = ccv_nnc_tensor_count(g->info);
	assert(tensor_count ==  ccv_nnc_tensor_count(a->info));
	assert(tensor_count ==  ccv_nnc_tensor_count(m->info));
	assert(tensor_count ==  ccv_nnc_tensor_count(v->info));
	assert(tensor_count ==  ccv_nnc_tensor_count(b->info));
	assert(tensor_count ==  ccv_nnc_tensor_count(n->info));
	assert(tensor_count ==  ccv_nnc_tensor_count(u->info));
	if (g->info.datatype == CCV_16F)
	{
		if (b->info.datatype == CCV_16F)
			_ccv_nnc_adamw_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, scale, beta1, beta2, rate_decay, rate_inv_bias_correction1, inv_bias_correction2, epsilon, (__half*)g->data.f16, (__half*)a->data.f16, (__half*)m->data.f16, (__half*)v->data.f16, (__half*)b->data.f16, (__half*)n->data.f16, (__half*)u->data.f16);
		else if (b->info.datatype == CCV_32F)
			_ccv_nnc_adamw_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, scale, beta1, beta2, rate_decay, rate_inv_bias_correction1, inv_bias_correction2, epsilon, (__half*)g->data.f16, a->data.f32, m->data.f32, v->data.f32, b->data.f32, n->data.f32, u->data.f32);
	} else if (g->info.datatype == CCV_32F) {
		if (b->info.datatype == CCV_16F)
			_ccv_nnc_adamw_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, scale, beta1, beta2, rate_decay, rate_inv_bias_correction1, inv_bias_correction2, epsilon, g->data.f32, (__half*)a->data.f16, (__half*)m->data.f16, (__half*)v->data.f16, (__half*)b->data.f16, (__half*)n->data.f16, (__half*)u->data.f16);
		else if (b->info.datatype == CCV_32F)
			_ccv_nnc_adamw_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, scale, beta1, beta2, rate_decay, rate_inv_bias_correction1, inv_bias_correction2, epsilon, g->data.f32, a->data.f32, m->data.f32, v->data.f32, b->data.f32, n->data.f32, u->data.f32);
	}
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_adamw_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	return CCV_NNC_EXEC_INVALID;
}

#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_ADAMW_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDA
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_adamw_forw;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_ADAMW_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDA
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_adamw_back;
#endif
}
