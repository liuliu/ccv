#include "hip/hip_runtime.h"
extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

#ifdef HAVE_CUDA

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_lamb_kernel(const size_t tensor_count, const float scale, const float beta1, const float beta2, const float decay, const float inv_bias_correction1, const float inv_bias_correction2, const float epsilon, const NUM1* const g, const NUM2* const a, const NUM2* const mom, const NUM2* const vel, float* const b, NUM2* const new_mom, NUM2* const new_vel)
{
	CUDA_1D_KERNEL_LOOP(i, tensor_count) {
		float grad = scale * (float)g[i];
		const float m = beta1 * (float)mom[i] + (1 - beta1) * grad;
		const float v = beta2 * (float)vel[i] + (1 - beta2) * grad * grad;
		b[i] = (m * inv_bias_correction1) / (sqrtf(v * inv_bias_correction2) + epsilon) + (float)a[i] * decay;
		new_mom[i] = (NUM2)m;
		new_vel[i] = (NUM2)v;
	}
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_lamb_kernel_to_float(const size_t tensor_count, const float scale, const float beta1, const float beta2, const float decay, const float inv_bias_correction1, const float inv_bias_correction2, const float epsilon, const NUM1* const g, const NUM2* const a, const NUM2* const mom, const NUM2* const vel, float* const b, float* const c, NUM2* const new_mom, NUM2* const new_vel)
{
	CUDA_1D_KERNEL_LOOP(i, tensor_count) {
		float grad = scale * (float)g[i];
		const float m = beta1 * (float)mom[i] + (1 - beta1) * grad;
		const float v = beta2 * (float)vel[i] + (1 - beta2) * grad * grad;
		b[i] = (m * inv_bias_correction1) / (sqrtf(v * inv_bias_correction2) + epsilon) + (float)a[i] * decay;
		c[i] = (float)a[i];
		new_mom[i] = (NUM2)m;
		new_vel[i] = (NUM2)v;
	}
}

__global__ void _ccv_nnc_rate_trust_ratio(const float rate, const float* const w_norm, const float* const update_norm, float* const rate_trust_ratio)
{
	const float trust_ratio = w_norm[0] > 0 && update_norm[0] > 0 ? w_norm[0] / update_norm[0] : 1;
	rate_trust_ratio[0] = rate * trust_ratio;
}

template<typename NUM>
__global__ void _ccv_nnc_lamb_kernel(const size_t tensor_count, const float* const rate_trust_ratio, const float* const g, const float* const a, NUM* const b)
{
	const float r = rate_trust_ratio[0];
	CUDA_1D_KERNEL_LOOP(i, tensor_count) {
		b[i] = (NUM)(a[i] - r * g[i]);
	}
}

static int _ccv_nnc_lamb_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 4);
	assert(output_size == 3);
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	hipblasHandle_t cublas = ccv_nnc_stream_context_get_cublas(stream_context);
	const int step = cmd.info.lamb.step;
	const float rate = cmd.info.lamb.rate;
	const float scale = cmd.info.lamb.scale;
	const float beta1 = cmd.info.lamb.beta1;
	const float beta2 = cmd.info.lamb.beta2;
	const float decay = cmd.info.lamb.decay;
	const float epsilon = cmd.info.lamb.epsilon;
	assert(step >= 1);
	const float inv_bias_correction1 = 1. / (1 - powf(beta1, step));
	const float inv_bias_correction2 = 1. / (1 - powf(beta2, step));
	assert(inputs[1]->info.datatype == inputs[2]->info.datatype &&
		inputs[2]->info.datatype == inputs[3]->info.datatype &&
		inputs[3]->info.datatype == outputs[0]->info.datatype &&
		outputs[0]->info.datatype == outputs[1]->info.datatype &&
		outputs[1]->info.datatype == outputs[2]->info.datatype);
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[0]));
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[1]));
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[2]));
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[3]));
	assert(CCV_IS_TENSOR_CONTIGUOUS(outputs[0]));
	assert(CCV_IS_TENSOR_CONTIGUOUS(outputs[1]));
	assert(CCV_IS_TENSOR_CONTIGUOUS(outputs[2]));
	const ccv_nnc_tensor_view_t* const g = (ccv_nnc_tensor_view_t*)inputs[0];
	const ccv_nnc_tensor_view_t* const a = (ccv_nnc_tensor_view_t*)inputs[1];
	const ccv_nnc_tensor_view_t* const m = (ccv_nnc_tensor_view_t*)inputs[2];
	const ccv_nnc_tensor_view_t* const v = (ccv_nnc_tensor_view_t*)inputs[3];
	ccv_nnc_tensor_view_t* const b = (ccv_nnc_tensor_view_t*)outputs[0];
	ccv_nnc_tensor_view_t* const n = (ccv_nnc_tensor_view_t*)outputs[1];
	ccv_nnc_tensor_view_t* const u = (ccv_nnc_tensor_view_t*)outputs[2];
	const size_t tensor_count = ccv_nnc_tensor_count(g->info);
	assert(tensor_count ==  ccv_nnc_tensor_count(a->info));
	assert(tensor_count ==  ccv_nnc_tensor_count(m->info));
	assert(tensor_count ==  ccv_nnc_tensor_count(v->info));
	assert(tensor_count ==  ccv_nnc_tensor_count(b->info));
	assert(tensor_count ==  ccv_nnc_tensor_count(n->info));
	assert(tensor_count ==  ccv_nnc_tensor_count(u->info));
	const size_t cublas_workspace_size = ccv_nnc_cublas_workspace_size_in_bytes(inputs, input_size, outputs, output_size);
	if (b->info.datatype == CCV_16F)
	{
		unsigned char* workspace = (unsigned char*)ccv_nnc_stream_context_get_workspace(stream_context, cublas_workspace_size + sizeof(float) * (tensor_count * 2 + 2), CCV_TENSOR_GPU_MEMORY);
		float* const update = (float*)(workspace + cublas_workspace_size);
		ccv_nnc_stream_context_set_cublas_workspace(cublas, stream_context, cublas_workspace_size);
		float* const af = update + tensor_count;
		if (g->info.datatype == CCV_16F)
		{
			_ccv_nnc_lamb_kernel_to_float<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, scale, beta1, beta2, decay, inv_bias_correction1, inv_bias_correction2, epsilon, (__half*)g->data.f16, (__half*)a->data.f16, (__half*)m->data.f16, (__half*)v->data.f16, update, af, (__half*)n->data.f16, (__half*)u->data.f16);
		} else if (g->info.datatype == CCV_32F) {
			_ccv_nnc_lamb_kernel_to_float<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, scale, beta1, beta2, decay, inv_bias_correction1, inv_bias_correction2, epsilon, g->data.f32, (__half*)a->data.f16, (__half*)m->data.f16, (__half*)v->data.f16, update, af, (__half*)n->data.f16, (__half*)u->data.f16);
		}
		float* const w_norm = af + tensor_count;
		float* const update_norm = w_norm + 1;
		hipblasPointerMode_t oldMode;
		hipblasGetPointerMode(cublas, &oldMode);
		hipblasSetPointerMode(cublas, HIPBLAS_POINTER_MODE_DEVICE);
		hipblasSnrm2(cublas, tensor_count, af, 1, w_norm);
		hipblasSnrm2(cublas, tensor_count, update, 1, update_norm);
		// Set it back.
		hipblasSetPointerMode(cublas, oldMode);
		float* const rate_trust_ratio = update_norm; // Reuse the memory.
		_ccv_nnc_rate_trust_ratio<<<1, 1, 0, stream>>>(rate, w_norm, update_norm, rate_trust_ratio);
		_ccv_nnc_lamb_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, rate_trust_ratio, update, af, (__half*)b->data.f16);
	} else if (b->info.datatype == CCV_32F) {
		unsigned char* workspace = (unsigned char*)ccv_nnc_stream_context_get_workspace(stream_context, cublas_workspace_size + sizeof(float) * (tensor_count + 2), CCV_TENSOR_GPU_MEMORY);
		float* const update = (float*)(workspace + cublas_workspace_size);
		ccv_nnc_stream_context_set_cublas_workspace(cublas, stream_context, cublas_workspace_size);
		if (g->info.datatype == CCV_16F)
		{
			_ccv_nnc_lamb_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, scale, beta1, beta2, decay, inv_bias_correction1, inv_bias_correction2, epsilon, (__half*)g->data.f16, a->data.f32, m->data.f32, v->data.f32, update, n->data.f32, u->data.f32);
		} else if (g->info.datatype == CCV_32F) {
			_ccv_nnc_lamb_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, scale, beta1, beta2, decay, inv_bias_correction1, inv_bias_correction2, epsilon, g->data.f32, a->data.f32, m->data.f32, v->data.f32, update, n->data.f32, u->data.f32);
		}
		float* const w_norm = update + tensor_count;
		float* const update_norm = w_norm + 1;
		hipblasPointerMode_t oldMode;
		hipblasGetPointerMode(cublas, &oldMode);
		hipblasSetPointerMode(cublas, HIPBLAS_POINTER_MODE_DEVICE);
		hipblasSnrm2(cublas, tensor_count, a->data.f32, 1, w_norm);
		hipblasSnrm2(cublas, tensor_count, update, 1, update_norm);
		// Set it back.
		hipblasSetPointerMode(cublas, oldMode);
		float* const rate_trust_ratio = update_norm; // Reuse the memory.
		_ccv_nnc_rate_trust_ratio<<<1, 1, 0, stream>>>(rate, w_norm, update_norm, rate_trust_ratio);
		_ccv_nnc_lamb_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, rate_trust_ratio, update, a->data.f32, b->data.f32);
	}
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_lamb_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	return CCV_NNC_EXEC_INVALID;
}

#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_LAMB_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDA
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_lamb_forw;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_LAMB_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDA
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_lamb_back;
#endif
}
