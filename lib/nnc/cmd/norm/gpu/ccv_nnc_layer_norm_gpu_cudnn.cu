#include "hip/hip_runtime.h"
extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

#ifdef HAVE_CUDNN

template<typename NUM>
__global__ void _ccv_nnc_inv_std_kernel(const int count, const float epsilon, const NUM* const a, NUM* const b)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		b[i] = (NUM)(1. / sqrt((float)a[i] * (float)a[i] + epsilon));
	}
}

static int _ccv_nnc_layer_norm_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 3);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	static const float one = 1, zero = 0, neg_one = -1;
	assert(output_size == 3);
	const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[1]));
	const ccv_nnc_cudnn_tensor_view_descriptor_t scale = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[1]);
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[2]));
	const ccv_nnc_cudnn_tensor_view_descriptor_t bias = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[2]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t b = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
	assert(CCV_IS_TENSOR_CONTIGUOUS(outputs[1]));
	const ccv_nnc_cudnn_tensor_view_descriptor_t saved_mean = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)outputs[1]);
	assert(CCV_IS_TENSOR_CONTIGUOUS(outputs[2]));
	assert(outputs[1]->info.datatype == outputs[2]->info.datatype);
	const int saved_datatype = outputs[1]->info.datatype;
	const ccv_nnc_cudnn_tensor_view_descriptor_t saved_inv_std = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)outputs[2]);
	int adim[CCV_NNC_MAX_DIM_ALLOC];
	int rdim[CCV_NNC_MAX_DIM_ALLOC];
	ccv_nnc_tensor_view_get_dim((ccv_nnc_tensor_view_t*)inputs[0], adim);
	ccv_nnc_tensor_view_get_dim((ccv_nnc_tensor_view_t*)outputs[1], rdim);
	assert(ccv_nnc_tensor_view_check_dim((ccv_nnc_tensor_view_t*)outputs[2], rdim));
	assert(ccv_nnc_tensor_view_check_dim((ccv_nnc_tensor_view_t*)outputs[0], adim));
	int x;
	int n = 1;
	for (x = 0; x < CCV_NNC_MAX_DIM + 2; x++)
		n *= adim[x];
	for (x = 0; x < CCV_NNC_MAX_DIM + 2; x++)
		n /= rdim[x];
	int rcount = 1;
	for (x = 0; x < CCV_NNC_MAX_DIM + 2; x++)
		rcount *= rdim[x];
	const float inv_n = 1. / n;
	hipdnnReduceTensorDescriptor_t reduce = ccv_nnc_stream_context_get_reduce_tensor_descriptor(stream_context);
	size_t saved_mean_workspace_size = 0;
	size_t saved_inv_std_workspace_size = 0;
	hipdnnSetReduceTensorDescriptor(reduce, HIPDNN_REDUCE_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN, HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES);
	CUDNN_ENFORCE(hipdnnGetReductionWorkspaceSize(cudnn, reduce, a.descriptor, saved_mean.descriptor, &saved_mean_workspace_size));
	hipdnnSetReduceTensorDescriptor(reduce, HIPDNN_REDUCE_TENSOR_NORM2, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN, HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES);
	CUDNN_ENFORCE(hipdnnGetReductionWorkspaceSize(cudnn, reduce, b.descriptor, saved_inv_std.descriptor, &saved_inv_std_workspace_size));
	const size_t workspace_size = ccv_max(saved_mean_workspace_size, saved_inv_std_workspace_size) + sizeof(float) * rcount;
	uint8_t* const workspace = (uint8_t*)ccv_nnc_stream_context_get_workspace(stream_context, workspace_size, CCV_TENSOR_GPU_MEMORY);
	hipdnnSetReduceTensorDescriptor(reduce, HIPDNN_REDUCE_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN, HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES);
	CUDNN_ENFORCE(hipdnnReduceTensor(cudnn, reduce, 0, 0, workspace, workspace_size, &inv_n, a.descriptor, a.data.u8, &zero, saved_mean.descriptor, saved_mean.data.u8));
	hipdnnOpTensorDescriptor_t op = ccv_nnc_stream_context_get_op_tensor_descriptor(stream_context);
	hipdnnSetOpTensorDescriptor(op, HIPDNN_OP_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN);
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, a.descriptor, a.data.u8, &neg_one, saved_mean.descriptor, saved_mean.data.u8, &zero, b.descriptor, b.data.u8));
	hipdnnSetReduceTensorDescriptor(reduce, HIPDNN_REDUCE_TENSOR_NORM2, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN, HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES);
	const float inv_n_sqrt = sqrt(inv_n);
	CUDNN_ENFORCE(hipdnnReduceTensor(cudnn, reduce, 0, 0, workspace, workspace_size, &inv_n_sqrt, b.descriptor, b.data.u8, &zero, saved_inv_std.descriptor, saved_inv_std.data.u8));
	// The epsilon is used a little bit differently from batch norm, it is outside of the sqrt in this case.
	const float epsilon = cmd.info.lnorm.epsilon;
	if (saved_datatype == CCV_32F)
		_ccv_nnc_inv_std_kernel<<<CUDA_GET_BLOCKS(rcount), CUDA_NUM_THREADS, 0, stream>>>(rcount, epsilon, saved_inv_std.data.f32, saved_inv_std.data.f32);
	else if (saved_datatype == CCV_16F)
		_ccv_nnc_inv_std_kernel<<<CUDA_GET_BLOCKS(rcount), CUDA_NUM_THREADS, 0, stream>>>(rcount, epsilon, (__half*)saved_inv_std.data.f16, (__half*)saved_inv_std.data.f16);
	hipdnnSetOpTensorDescriptor(op, HIPDNN_OP_TENSOR_MUL, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN);
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, b.descriptor, b.data.u8, &one, saved_inv_std.descriptor, saved_inv_std.data.u8, &zero, b.descriptor, b.data.u8));
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, b.descriptor, b.data.u8, &one, scale.descriptor, scale.data.u8, &zero, b.descriptor, b.data.u8));
	hipdnnSetOpTensorDescriptor(op, HIPDNN_OP_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN);
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, b.descriptor, b.data.u8, &one, bias.descriptor, bias.data.u8, &zero, b.descriptor, b.data.u8));
	ccv_nnc_stream_context_return_reduce_tensor_descriptor(stream_context, reduce);
	ccv_nnc_stream_context_return_op_tensor_descriptor(stream_context, op);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(b);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(scale);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(bias);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(saved_mean);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(saved_inv_std);
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_layer_norm_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 9);
	assert(output_size >= 1);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	const ccv_nnc_cudnn_tensor_view_descriptor_t g = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[3]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t h = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[4]));
	const ccv_nnc_cudnn_tensor_view_descriptor_t scale = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[4]);
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[7]));
	const ccv_nnc_cudnn_tensor_view_descriptor_t saved_mean = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[7]);
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[8]));
	const ccv_nnc_cudnn_tensor_view_descriptor_t saved_inv_std = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[8]);
	if (output_size > 1 && outputs[1])
		{ assert(CCV_IS_TENSOR_CONTIGUOUS(outputs[1])); }
	const ccv_nnc_cudnn_tensor_view_descriptor_t dscale = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, output_size > 1 ? (const ccv_nnc_tensor_view_t*)outputs[1] : 0);
	if (output_size > 2 && outputs[2])
		{ assert(CCV_IS_TENSOR_CONTIGUOUS(outputs[2])); }
	const ccv_nnc_cudnn_tensor_view_descriptor_t dbias = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, output_size > 2 ? (const ccv_nnc_tensor_view_t*)outputs[2] : 0);
	int gdim[CCV_NNC_MAX_DIM_ALLOC];
	int rdim[CCV_NNC_MAX_DIM_ALLOC];
	ccv_nnc_tensor_view_get_dim((ccv_nnc_tensor_view_t*)inputs[0], gdim);
	ccv_nnc_tensor_view_get_dim((ccv_nnc_tensor_view_t*)inputs[7], rdim);
	assert(ccv_nnc_tensor_view_check_dim((ccv_nnc_tensor_view_t*)inputs[8], rdim));
	assert(ccv_nnc_tensor_view_check_dim((ccv_nnc_tensor_view_t*)inputs[3], gdim));
	assert(ccv_nnc_tensor_view_check_dim((ccv_nnc_tensor_view_t*)outputs[0], gdim));
	static const float one = 1, zero = 0, neg_one = -1;
	int x;
	int n = 1;
	for (x = 0; x < CCV_NNC_MAX_DIM + 2; x++)
		n *= gdim[x];
	for (x = 0; x < CCV_NNC_MAX_DIM + 2; x++)
		n /= rdim[x];
	int gcount = 1, rcount = 1;
	for (x = 0; x < CCV_NNC_MAX_DIM + 2; x++)
		gcount *= gdim[x], rcount *= rdim[x];
	const float neg_inv_n = -1. / n;
	hipdnnReduceTensorDescriptor_t reduce = ccv_nnc_stream_context_get_reduce_tensor_descriptor(stream_context);
	hipdnnSetReduceTensorDescriptor(reduce, HIPDNN_REDUCE_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN, HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES);
	size_t scale_workspace_size = 0;
	if (dscale.descriptor)
		{ CUDNN_ENFORCE(hipdnnGetReductionWorkspaceSize(cudnn, reduce, g.descriptor, dscale.descriptor, &scale_workspace_size)); }
	size_t mean_workspace_size = 0;
	CUDNN_ENFORCE(hipdnnGetReductionWorkspaceSize(cudnn, reduce, g.descriptor, saved_mean.descriptor, &mean_workspace_size));
	const size_t workspace_size = ccv_max(scale_workspace_size, mean_workspace_size);
	uint8_t* const workspace = (uint8_t*)ccv_nnc_stream_context_get_workspace(stream_context, workspace_size + sizeof(float) * gcount * 3 + sizeof(float) * rcount * 2, CCV_TENSOR_GPU_MEMORY);
	if (dbias.descriptor)
		{ CUDNN_ENFORCE(hipdnnReduceTensor(cudnn, reduce, 0, 0, workspace, workspace_size, &one, g.descriptor, g.data.u8, &zero, dbias.descriptor, dbias.data.u8)); }
	float* const ahp = (float*)(workspace + workspace_size);
	const ccv_nnc_tensor_t aht = ccv_nnc_tensor(ahp, inputs[0]->info, 0);
	const ccv_nnc_cudnn_tensor_view_descriptor_t ah = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)&aht);
	float* const gssp = ahp + gcount;
	const ccv_nnc_tensor_t gsst = ccv_nnc_tensor(gssp, inputs[0]->info, 0);
	const ccv_nnc_cudnn_tensor_view_descriptor_t gss = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)&gsst);
	float* const ahgssp = gssp + gcount;
	const ccv_nnc_tensor_t ahgsst = ccv_nnc_tensor(ahgssp, inputs[0]->info, 0);
	const ccv_nnc_cudnn_tensor_view_descriptor_t ahgss = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)&ahgsst);
	float* const gssrp = ahgssp + gcount;
	const ccv_nnc_tensor_t gssrt = ccv_nnc_tensor(gssrp, inputs[7]->info, 0);
	const ccv_nnc_cudnn_tensor_view_descriptor_t gssr = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)&gssrt);
	float* const ahgssrp = gssrp + rcount;
	const ccv_nnc_tensor_t ahgssrt = ccv_nnc_tensor(ahgssrp, inputs[7]->info, 0);
	const ccv_nnc_cudnn_tensor_view_descriptor_t ahgssr = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)&ahgssrt);
	hipdnnOpTensorDescriptor_t op = ccv_nnc_stream_context_get_op_tensor_descriptor(stream_context);
	hipdnnSetOpTensorDescriptor(op, HIPDNN_OP_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN);
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, a.descriptor, a.data.u8, &neg_one, saved_mean.descriptor, saved_mean.data.u8, &zero, ah.descriptor, ah.data.u8));
	hipdnnSetOpTensorDescriptor(op, HIPDNN_OP_TENSOR_MUL, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN);
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, ah.descriptor, ah.data.u8, &one, saved_inv_std.descriptor, saved_inv_std.data.u8, &zero, ah.descriptor, ah.data.u8));
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, ah.descriptor, ah.data.u8, &one, g.descriptor, g.data.u8, &zero, ahgss.descriptor, ahgss.data.u8));
	if (dscale.descriptor)
		{ CUDNN_ENFORCE(hipdnnReduceTensor(cudnn, reduce, 0, 0, workspace, workspace_size, &one, ahgss.descriptor, ahgss.data.u8, &zero, dscale.descriptor, dscale.data.u8)); }
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, g.descriptor, g.data.u8, &one, scale.descriptor, scale.data.u8, &zero, gss.descriptor, gss.data.u8));
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, gss.descriptor, gss.data.u8, &one, saved_inv_std.descriptor, saved_inv_std.data.u8, &zero, gss.descriptor, gss.data.u8));
	CUDNN_ENFORCE(hipdnnReduceTensor(cudnn, reduce, 0, 0, workspace, workspace_size, &one, gss.descriptor, gss.data.u8, &zero, gssr.descriptor, gssr.data.u8));
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, ah.descriptor, ah.data.u8, &one, gss.descriptor, gss.data.u8, &zero, ahgss.descriptor, ahgss.data.u8));
	CUDNN_ENFORCE(hipdnnReduceTensor(cudnn, reduce, 0, 0, workspace, workspace_size, &one, ahgss.descriptor, ahgss.data.u8, &zero, ahgssr.descriptor, ahgssr.data.u8));
	ccv_nnc_stream_context_return_reduce_tensor_descriptor(stream_context, reduce);
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, ah.descriptor, ah.data.u8, &one, ahgssr.descriptor, ahgssr.data.u8, &zero, ah.descriptor, ah.data.u8));
	hipdnnSetOpTensorDescriptor(op, HIPDNN_OP_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN);
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, ah.descriptor, ah.data.u8, &one, gssr.descriptor, gssr.data.u8, &zero, ah.descriptor, ah.data.u8));
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, gss.descriptor, gss.data.u8, &neg_inv_n, ah.descriptor, ah.data.u8, &zero, h.descriptor, h.data.u8));
	ccv_nnc_stream_context_return_op_tensor_descriptor(stream_context, op);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(g);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(h);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(scale);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(saved_mean);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(saved_inv_std);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(dscale);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(dbias);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(ah);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(gss);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(ahgss);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(gssr);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(ahgssr);
	return CCV_NNC_EXEC_SUCCESS;
}

#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_LAYER_NORM_FORWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_layer_norm_forw;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_LAYER_NORM_BACKWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_layer_norm_back;
#endif
}
